#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel. Each thread takes care of one element of c
__global__ void hello_cuda() {
  printf("hello from the GPU\n");
}
 
int main( int argc, char* argv[] )
{
  // Execute the kernel
  hello_cuda<<<1, 1>>>();

  hipDeviceSynchronize(); 

  return 0;
}
